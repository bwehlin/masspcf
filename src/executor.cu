#include <mpcf/executor.h>
#include <taskflow/taskflow.hpp>

#include <stdexcept>

#ifdef BUILD_WITH_CUDA
#pragma message("Building mpcf_cpp with CUDA")
#else
#pragma message("Building mpcf_cpp without CUDA")
#endif

#ifdef BUILD_WITH_CUDA
#include <hip/hip_runtime.h>
#include <mpcf/cuda/cuda_util.cuh>
#endif

size_t mpcf::get_num_cuda_devices()
{
  int nGpus = 0;
#ifdef BUILD_WITH_CUDA
  if (hipGetDeviceCount(&nGpus) != hipSuccess)
  {
    return 0;
  }
  if (nGpus < 0)
  {
    // Just in case...
    throw std::runtime_error("Negative number (" + std::to_string(nGpus) + ") of GPUs reported!");
  }
#endif
  return static_cast<size_t>(nGpus);
}

mpcf::Executor& mpcf::default_executor()
{
  static Executor exec = Executor(std::thread::hardware_concurrency(), get_num_cuda_devices());
  return exec;
}
