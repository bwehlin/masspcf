#include "algorithms/cuda_matrix_integrate.h"

#include <hip/hip_runtime.h>
#include <>

#include <iostream>

namespace
{
  template <typename T>
  __device__
  float l1_inner_prod(T, T, T t, T b)
  {
    return t * b;
  }
  
  __device__
  float l1_inner_prod_f32_impl(float l, float r, float t, float b)
  {
    return l1_inner_prod<float>(l, r, t, b);
  }
  
  __device__
  double l1_inner_prod_f64_impl(double l, double r, double t, double b)
  {
    return l1_inner_prod<double>(l, r, t, b);
  }
}

mpcf::DeviceOp<float, float>
mpcf::device_ops::l1_inner_prod_f32()
{
  return &l1_inner_prod_f32_impl;
}

mpcf::DeviceOp<double, double>
mpcf::device_ops::l1_inner_prod_f64()
{
  return &l1_inner_prod_f64_impl;
}

void
mpcf::detail::cuda_matrix_integrate_f32(float* out, const std::vector<Pcf_f32>& fs, DeviceOp<float, float> op)
{
  std::cout << "Hello" << std::endl;
}

void
mpcf::detail::cuda_matrix_integrate_f64(double* out, const std::vector<Pcf_f64>& fs, DeviceOp<double, double> op)
{
  std::cout << "Hello64" << std::endl;
}
